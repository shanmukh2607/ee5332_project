#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include<sys/time.h>


__global__ void fir(int *in, int *kernel, int *out, int I, int K){
    __shared__ int Acc[17];
    Acc[threadIdx.x] = 0;
    int temp;
    int w = kernel[threadIdx.x];
    for(int i=0; i<I+K-1; i++){
        temp = in[i]*w + Acc[threadIdx.x];
        //if(threadIdx.x ==1) printf("%d %d %d\n", temp,in[i],w);
        //__syncthreads(); not required since #threads < warpsize=32
        Acc[threadIdx.x+1] = temp;
        if(threadIdx.x == 15) out[i] = temp;
        __syncthreads();
    }
}

void readMatrix(FILE *inputFilePtr, int *matrix, int rows, int cols) {
	for(int i=0; i<rows; i++) {
		for(int j=0; j<cols; j++) {
			fscanf(inputFilePtr, "%d", &matrix[i*cols+j]);
		}
	}
}

void writeMatrix(FILE *outputFilePtr, int *matrix, int ksize, int isize) {
	
		for(int j=0; j<isize; j++) {
			fprintf(outputFilePtr, "%d\n", matrix[ksize-1 +j]);
		}
		fprintf(outputFilePtr, "\n");
}



int main(){
    int *image, *kernel, *conv;
    int ISIZE,KSIZE;
    // FILE POINTERS
    FILE *imageFilePtr, *kernelFilePtr, *outputFilePtr;
    
    imageFilePtr = fopen("1Dinput.dat", "r");
	if(imageFilePtr == NULL) {
	    printf("Failed to open the image file.!!\n"); 
		return 0;
	}

    kernelFilePtr = fopen("1Dkernel.dat", "r");
	if(kernelFilePtr == NULL) {
	    printf("Failed to open the kernel file.!!\n"); 
		return 0;
	}
    // reads ISIZE
    fscanf(imageFilePtr, "%d",&ISIZE);
    // reads KSIZE
    fscanf(kernelFilePtr, "%d", &KSIZE);

    // allocate memory
    image = (int *)malloc((ISIZE+KSIZE-1)*sizeof(int));
    kernel = (int *)malloc(KSIZE*sizeof(int));
	//acc = (int *)malloc(KSIZE*sizeof(int));
    conv = (int *)malloc((ISIZE+KSIZE-1)*sizeof(int));
	memset(image,0,(ISIZE+KSIZE-1)*sizeof(int));
	//memset(acc,0,KSIZE*sizeof(int));
    // File io
    // read image and filter data
    readMatrix(imageFilePtr, image, 1, ISIZE);
	readMatrix(kernelFilePtr, kernel, 1, KSIZE);

    //=============================================================================
     hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;
    hipEventRecord(start,0);

    // device arrays declaration

    int *dinput, *dkernel, *dconv;
    hipMalloc(&dinput, (ISIZE+KSIZE-1)*sizeof(int));
    hipMalloc(&dkernel, KSIZE*sizeof(int));
    hipMalloc(&dconv, (ISIZE+KSIZE-1)*sizeof(int));

    // CUDA Memcpys
    hipMemcpy(dinput, image, (ISIZE+KSIZE-1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dkernel, kernel, KSIZE*sizeof(int), hipMemcpyHostToDevice);
    // kernel call
    fir<<<1,16>>>(dinput,dkernel,dconv,ISIZE,KSIZE);
    hipMemcpy(conv,dconv,(ISIZE+KSIZE-1)*sizeof(int), hipMemcpyDeviceToHost);
    hipFree(dinput);
    hipFree(dkernel);
    hipFree(dconv);

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time taken by function to execute is: %.6f ms\n", milliseconds);
    //======================================================================================
    


    // Write output
    outputFilePtr = fopen("1Doutputgolden.dat", "w");
	writeMatrix(outputFilePtr, conv, KSIZE, ISIZE);

    // close files
    fclose(imageFilePtr);
    fclose(kernelFilePtr);
    fclose(outputFilePtr);

	// deallocate memory
	free(image);
    free(kernel);
    free(conv);
    return 0;
}